#include "hip/hip_runtime.h"
/*
* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix_world.h>
#include "random.h"
#include "Packet.h"

rtDeclareVariable(optix::float3, bg_color, , );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );


rtDeclareVariable(PerRayData_pathtrace_shadow, prd_shadow, rtPayload, );
rtDeclareVariable(PerRayData_pathtrace, prd_path, rtPayload, );

/*RT_PROGRAM void miss()
{
prd_path.result = bg_color;
}
*/

rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void miss()
{
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	prd_path.radiance = make_float3(0);// tex2D(envmap, u, v));
	prd_path.done = true;
	prd_path.depth = 10000.0f;

}

RT_PROGRAM void missShadow()
{
	// this material is opaque, so it fully attenuates all shadow rays
	prd_shadow.inShadow = false;
}
